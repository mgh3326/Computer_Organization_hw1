#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#define N 1000	//  number of threads per block

#define T 1// number of block


__global__ void add(int a, int b, int *c) {
	*c = a + b;
}
int main(int argc, char **argv) {

	int a, b, c;
	int *dev_c;
	a = 3;
	b = 4;
	hipMalloc((void**)&dev_c, sizeof(int));
	add << <1, 1 >> > (a, b, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int),
		hipMemcpyDeviceToHost);
	printf("%d + %d is %d\n", a, b, c);
	hipFree(dev_c);
	return 0;
}


// Helper function for using CUDA to add vectors in parallel.

