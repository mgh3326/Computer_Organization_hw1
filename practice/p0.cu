#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#define N 1000	//  number of threads per block

#define T 10000// number of block


__global__ void vecAdd(int *A, int *B, int *C) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	C[i] = A[i] * 10 + B[i];
}
int main(int argc, char **argv) {

	int size = N * T * sizeof(int);
	int *devA, *devB, *devC;
	int  *a, *b, *c;

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);
	hipMalloc((void**)&devA, size);
	hipMalloc((void**)&devB, size);
	hipMalloc((void**)&devC, size);
	for (int i = 0; i < N*T; i++) {
		a[i] = i;
		b[i] = 1;
	}
	hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, size, hipMemcpyHostToDevice);
	vecAdd << <T, N >> > (devA, devB, devC);
	hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	for (int i = 0; i < N*T; i++) {
		printf("%d = %d * 10 * %d\n", c[i], a[i], b[i]);
	}
	
	return(0);
}
// Helper function for using CUDA to add vectors in parallel.

